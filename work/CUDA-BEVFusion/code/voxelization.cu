#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_fp16.h>

#include "common/check.hpp"
#include "common/launch.cuh"
#include "lidar-voxelization.hpp"

namespace bevfusion {
namespace lidar {

static __device__ inline uint64_t hash(uint64_t k) {
  k ^= k >> 16;
  k *= 0x85ebca6b;
  k ^= k >> 13;
  k *= 0xc2b2ae35;
  k ^= k >> 16;
  return k;
}

static __device__ inline void insert_to_hash_table(const uint32_t key, uint32_t *value, const uint32_t hash_size,
                                                   uint32_t *hash_table) {
  uint64_t hash_value = hash(key);// 获取hash值
  uint32_t slot = hash_value % (hash_size / 2) /*key, value*/;//获取 实际点云数量*2 的余数 作为偏移量 
  uint32_t empty_key = UINT32_MAX;
  while (true) {
    uint32_t pre_key = atomicCAS(hash_table + slot, empty_key, key); //((hash_table + slot) == compare ? key : hash_table + slot)
    if (pre_key == empty_key) {
      hash_table[slot + hash_size / 2 /*offset*/] = atomicAdd(value, 1); // 在hashtable中对应位置添加atomicAdd(value, 1) 表示第几个voxel
      break;
    } else if (pre_key == key) {
      break;
    }
    slot = (slot + 1) % (hash_size / 2);
  }
}

static __device__ inline uint32_t lookup_hash_table(const uint32_t key, const uint32_t hash_size, const uint32_t *hash_table) {
  uint64_t hash_value = hash(key);
  uint32_t slot = hash_value % (hash_size / 2) /*key, value*/;
  uint32_t empty_key = UINT32_MAX;
  int cnt = 0;
  while (true /* need to be adjusted according to data*/) {
    cnt++;
    if (hash_table[slot] == key) {
      return hash_table[slot + hash_size / 2];
    } else if (hash_table[slot] == empty_key) {
      return empty_key;
    } else {
      slot = (slot + 1) % (hash_size / 2);
    }
  }
  return empty_key;
}

static __global__ void build_hash_table_kernel(size_t points_size, const half *points, VoxelizationParameter param,
                                               unsigned int *hash_table, unsigned int *real_voxel_num) {
  int point_idx = cuda_linear_index;
  if (point_idx >= points_size) return;
  // 获取点云的xyz值
   float px = points[param.num_feature * point_idx];
  float py = points[param.num_feature * point_idx + 1];
  float pz = points[param.num_feature * point_idx + 2];
  // 转换为voxel的xyz index
  int voxel_idx = floorf((px - param.min_range.x) / param.voxel_size.x);
  if (voxel_idx < 0 || voxel_idx >= param.grid_size.x) return;

  int voxel_idy = floorf((py - param.min_range.y) / param.voxel_size.y);
  if (voxel_idy < 0 || voxel_idy >= param.grid_size.y) return;

  int voxel_idz = floorf((pz - param.min_range.z) / param.voxel_size.z);
  if (voxel_idz < 0 || voxel_idz >= param.grid_size.z) return;
  // 计算voxel的一维 index
  unsigned int voxel_offset = (voxel_idz * param.grid_size.y + voxel_idy) * param.grid_size.x + voxel_idx;
  insert_to_hash_table(voxel_offset, real_voxel_num, points_size * 2 * 2, hash_table);// points_size 当前帧点云数量
}

template <CoordinateOrder order>
static __device__ void save_result_by_order(uint4 *output, uint x, uint y, uint z);

template <>
__device__ void save_result_by_order<CoordinateOrder::XYZ>(uint4 *output, uint x, uint y, uint z) {
  *output = make_uint4(0, x, y, z);
}

template <>
__device__ void save_result_by_order<CoordinateOrder::ZYX>(uint4 *output, uint x, uint y, uint z) {
  *output = make_uint4(0, z, y, x);
}

template <CoordinateOrder order>
static __global__ void voxelization_kernel(size_t points_size, const half *points, VoxelizationParameter param,
                                           unsigned int *hash_table, unsigned int *num_points_per_voxel, float *voxels_temp,
                                           unsigned int *voxel_indices) {
  int point_idx = cuda_linear_index;
  if (point_idx >= points_size) return;

  float px = points[param.num_feature * point_idx];// 点云坐标x，单位m
  float py = points[param.num_feature * point_idx + 1];// 点云坐标y，单位m
  float pz = points[param.num_feature * point_idx + 2];// 点云坐标z，单位m

  if (px < param.min_range.x || px >= param.max_range.x || py < param.min_range.y || py >= param.max_range.y ||
      pz < param.min_range.z || pz >= param.max_range.z) {
    return;
  }
  // 转换为voxel index xyz
  int voxel_idx = floorf((px - param.min_range.x) / param.voxel_size.x);
  int voxel_idy = floorf((py - param.min_range.y) / param.voxel_size.y);
  int voxel_idz = floorf((pz - param.min_range.z) / param.voxel_size.z);
  if ((voxel_idx < 0 || voxel_idx >= param.grid_size.x)) {
    return;
  }
  if ((voxel_idy < 0 || voxel_idy >= param.grid_size.y)) {
    return;
  }
  if ((voxel_idz < 0 || voxel_idz >= param.grid_size.z)) {
    return;
  }

  unsigned int voxel_offset = (voxel_idz * param.grid_size.y + voxel_idy) * param.grid_size.x + voxel_idx; // 改点对应voxel在一维上的index

  // scatter to voxels
  // 在voxel_offset key对应hash地址中记录该voxel收纳了多少个point，超过10个话直接返回
  unsigned int voxel_id = lookup_hash_table(voxel_offset, points_size * 2 * 2, hash_table);
  if (voxel_id >= param.max_voxels) {
    return;
  }
  // 在featuremap的voxels_temp 插入该voxel下的点
  unsigned int current_num = atomicAdd(num_points_per_voxel + voxel_id, 1);
  if (current_num < param.max_points_per_voxel) {
    unsigned int dst_offset = voxel_id * (param.num_feature * param.max_points_per_voxel) + current_num * param.num_feature;
    unsigned int src_offset = point_idx * param.num_feature;
    for (int feature_idx = 0; feature_idx < param.num_feature; ++feature_idx) {
      voxels_temp[dst_offset + feature_idx] = points[src_offset + feature_idx];
    }

    // now only deal with batch_size = 1
    // since not sure what the input format will be if batch size > 1
    save_result_by_order<order>(&((uint4 *)voxel_indices)[voxel_id], voxel_idx, voxel_idy, voxel_idz);
  }
}

static __global__ void reduce_mean_kernel(size_t num_voxels, float *voxels_temp, unsigned int *num_points_per_voxel,
                                          int max_points_per_voxel, int feature_num, half *voxel_features) {
  int voxel_idx = cuda_linear_index;
  if (voxel_idx >= num_voxels) return;

  num_points_per_voxel[voxel_idx] =
      num_points_per_voxel[voxel_idx] > max_points_per_voxel ? max_points_per_voxel : num_points_per_voxel[voxel_idx];
  int valid_points_num = num_points_per_voxel[voxel_idx];
  int offset = voxel_idx * max_points_per_voxel * feature_num;
  // 各voxel第一个点的xyz替换成该voxel下所有有效点的平均值
  for (int feature_idx = 0; feature_idx < feature_num; ++feature_idx) {
    for (int point_idx = 0; point_idx < valid_points_num - 1; ++point_idx) {
      voxels_temp[offset + feature_idx] += voxels_temp[offset + (point_idx + 1) * feature_num + feature_idx];
    }
    voxels_temp[offset + feature_idx] /= valid_points_num;
  }

  // move to be continuous 只获取均值转换成fp16，连续储存
  for (int feature_idx = 0; feature_idx < feature_num; ++feature_idx) {
    int dst_offset = voxel_idx * feature_num;
    int src_offset = voxel_idx * feature_num * max_points_per_voxel;
    voxel_features[dst_offset + feature_idx] = __float2half(voxels_temp[src_offset + feature_idx]);
  }
}

nvtype::Int3 VoxelizationParameter::compute_grid_size(const nvtype::Float3 &max_range, const nvtype::Float3 &min_range,
                                                      const nvtype::Float3 &voxel_size) {
  nvtype::Int3 size;
  size.x = static_cast<int>(std::round((max_range.x - min_range.x) / voxel_size.x));
  size.y = static_cast<int>(std::round((max_range.y - min_range.y) / voxel_size.y));
  size.z = static_cast<int>(std::round((max_range.z - min_range.z) / voxel_size.z));
  return size;
}

class VoxelizationImplement : public Voxelization {
 public:
  virtual ~VoxelizationImplement() {
    if (hash_table_) checkRuntime(hipFree(hash_table_));
    if (voxels_temp_) checkRuntime(hipFree(voxels_temp_));

    if (d_voxel_features_) checkRuntime(hipFree(d_voxel_features_));
    if (d_voxel_num_) checkRuntime(hipFree(d_voxel_num_));
    if (d_voxel_indices_) checkRuntime(hipFree(d_voxel_indices_));

    if (d_real_num_voxels_) checkRuntime(hipFree(d_real_num_voxels_));
    if (h_real_num_voxels_) checkRuntime(hipHostFree(h_real_num_voxels_));
  }

  //分配device memory,这里可以参考一下CUDA-CenterPoint
  bool init(VoxelizationParameter param) {
    this->param_ = param;
    this->output_grid_size_ = {(int)param_.grid_size.x, (int)param_.grid_size.y, (int)param_.grid_size.z + 1}; //输出grid xyz尺寸

    this->hash_table_size_ = param_.max_points * 2 * 2 * sizeof(unsigned int);//hash_table_size_=300000*2*2*4
    this->voxels_temp_size_ = param_.max_voxels * param_.max_points_per_voxel * param_.num_feature * sizeof(float);// voxels_temp_size_= 160000*10*5*4
    this->voxel_features_size_ = param_.max_voxels * param_.max_points_per_voxel * param_.num_feature * sizeof(half);// 160000*10*5*2
    this->voxel_num_size_ = param_.max_voxels * sizeof(unsigned int);//160000*4
    this->voxel_idxs_size_ = param_.max_voxels * 4 * sizeof(unsigned int);//160000*4*4

    checkRuntime(hipMalloc(&hash_table_, hash_table_size_));
    checkRuntime(hipMalloc(&voxels_temp_, voxels_temp_size_));
    checkRuntime(hipMalloc(&d_voxel_features_, voxel_features_size_));
    checkRuntime(hipMalloc(&d_voxel_num_, voxel_num_size_));
    checkRuntime(hipMalloc(&d_voxel_indices_, voxel_idxs_size_));
    checkRuntime(hipMalloc(&d_real_num_voxels_, sizeof(unsigned int)));
    checkRuntime(hipHostMalloc(&h_real_num_voxels_, sizeof(unsigned int)));
    return true;
  }

  // points and voxels must be of half type
  // 这里面的实现可以参考一下CUDA-CenterPoint
  virtual void forward(const nvtype::half *points, int num_points, void *stream, CoordinateOrder output_order) override {
    hipStream_t _stream = reinterpret_cast<hipStream_t>(stream);
    const half *_points = reinterpret_cast<const half *>(points);
    checkRuntime(hipMemsetAsync(hash_table_, 0xff, hash_table_size_, _stream));
    checkRuntime(hipMemsetAsync(voxels_temp_, 0xff, voxels_temp_size_, _stream));
    checkRuntime(hipMemsetAsync(d_voxel_num_, 0, voxel_num_size_, _stream));
    checkRuntime(hipMemsetAsync(d_real_num_voxels_, 0, sizeof(unsigned int), _stream));
    // 通过核函数构建hashtabel，因为同一个voxel可能会存在多个点云
    cuda_linear_launch(build_hash_table_kernel, //核函数
                        _stream,                
                        num_points,             //当前帧lidarpoints数量，对应kernel启动数量
                         _points,               //点云数量
                         param_,                //参数
                         hash_table_,           //hashtabel地址
                         d_real_num_voxels_);   //记录实际使用voxel数量，初始化为0
    checkRuntime(hipMemcpyAsync(h_real_num_voxels_, d_real_num_voxels_, sizeof(int), hipMemcpyDeviceToHost, _stream));

    // for difference output order  将对应的feature选择前10个插入到featuremap中
    if (output_order == CoordinateOrder::XYZ) {
      cuda_linear_launch(voxelization_kernel<CoordinateOrder::XYZ>, _stream, num_points, _points, param_, hash_table_,
                         d_voxel_num_, voxels_temp_, d_voxel_indices_);
      this->output_grid_size_ = {(int)param_.grid_size.x, (int)param_.grid_size.y, (int)param_.grid_size.z + 1};
    } else if (output_order == CoordinateOrder::ZYX) {
      cuda_linear_launch(voxelization_kernel<CoordinateOrder::ZYX>, _stream, num_points, _points, param_, hash_table_,
                         d_voxel_num_, voxels_temp_, d_voxel_indices_);
      this->output_grid_size_ = {(int)param_.grid_size.z + 1, (int)param_.grid_size.y, (int)param_.grid_size.x};
    } else
      Assertf(false, "Invalid output_order: %d", static_cast<int>(output_order));

    checkRuntime(hipStreamSynchronize(_stream));
    // 取单个voxel内所有有效点的平均值
    real_num_voxels_ = *h_real_num_voxels_;
    cuda_linear_launch(reduce_mean_kernel, _stream, real_num_voxels_, voxels_temp_, d_voxel_num_, param_.max_points_per_voxel,
                       param_.num_feature, d_voxel_features_);
  }

  virtual unsigned int num_voxels() override { return real_num_voxels_; }

  virtual unsigned int voxel_dim() override { return param_.num_feature; }

  virtual unsigned int indices_dim() override { return 4; }

  virtual std::vector<int> grid_size() override { return output_grid_size_; }

  virtual const void *indices() override { return d_voxel_indices_; }

  virtual const void *features() override { return d_voxel_features_; }

  virtual CoordinateOrder order() override { return order_; }

 private:
  CoordinateOrder order_ = CoordinateOrder::NoneOrder;
  VoxelizationParameter param_;
  unsigned int real_num_voxels_ = 0;
  std::vector<int> output_grid_size_;

  unsigned int *hash_table_ = nullptr;
  float *voxels_temp_ = nullptr;
  unsigned int *d_real_num_voxels_ = nullptr;
  unsigned int *h_real_num_voxels_ = nullptr;
  unsigned int *d_voxel_num_ = nullptr;
  half *d_voxel_features_ = nullptr;
  unsigned int *d_voxel_indices_ = nullptr;
  unsigned int hash_table_size_;
  unsigned int voxels_temp_size_;
  unsigned int voxel_features_size_;
  unsigned int voxel_idxs_size_;
  unsigned int voxel_num_size_;
};

// 调用lidar命名空间接口下的一个实现类: VoxelizationImplement
// 主要用来负责将点云进行体素化
std::shared_ptr<Voxelization> create_voxelization(VoxelizationParameter param) {
  std::shared_ptr<VoxelizationImplement> impl(new VoxelizationImplement());
  if (!impl->init(param)) {
    impl.reset();
  }
  return impl;
}

};  // namespace lidar
};  // namespace bevfusion
